#include "hip/hip_runtime.h"
//
// Created by Fx Kaze on 25-1-6.
//

#include <hip/hip_runtime_api.h>

#include "CudaCalc.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <unistd.h>
#include "private/vars.h"


// #include "hip/hip_runtime.h"
// #include <highgui.hpp>

using namespace cv;

#define DIM 600   //图像长宽

__global__ void kernel(unsigned char* ptr)
{
	// map from blockIdx to pixel position
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int boffset = bx + by * gridDim.x;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int toffset = tx + ty * blockDim.x;
	int offset = toffset + boffset * blockDim.x * blockDim.y;
	// printf("offset: %d\n", offset);

	//BGR设置
	ptr[offset * 3 + 0] = 999 * tx * ty % 255;
	ptr[offset * 3 + 1] = 99 * tx * tx * ty * ty % 255;
	ptr[offset * 3 + 2] = 9 * offset * offset % 255;
}

int main()
{
	hipError_t error;
	Mat image = Mat(DIM, DIM, CV_8UC3, Scalar::all(0));
	u8* dev_bitmap;
	error = hipMalloc(&dev_bitmap, 3 * image.cols * image.rows);
	if (error != hipSuccess)
		printf("hipMalloc failed\n");

	dim3 block(60, 60);
	dim3 thread(10, 10);
	//DIM*DIM个线程块
	kernel<<<block, thread>>>(dev_bitmap);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("CUDA Error: %s\n", hipGetErrorString(err));
		// Possibly: exit(-1) if program cannot continue....
	}
	hipDeviceSynchronize();

	error = hipMemcpy(image.data, dev_bitmap,
	                   3 * image.cols * image.rows,
	                   hipMemcpyDeviceToHost);
	if (error != hipSuccess)
		printf("hipMemcpyDeviceToHost failed\n");

	hipFree(dev_bitmap);

	imshow("CUDA Grid/Block/Thread)", image);
	waitKey();
}

int __main()
{
	chdir(homePath);
	int devCount = 0;
	hipGetDeviceCount(&devCount);
	printf("Device count: %d\n", devCount);
	int devId = devCount - 1;
	hipSetDevice(devId);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devId);
	printf("Device name: %s\n", prop.name);
	printf("Device major: %d\n", prop.major);
	printf("Device minor: %d\n", prop.minor);
	printf("Total memory size: %lu\n", prop.totalGlobalMem);
	printf("Warp size: %d\n", prop.warpSize);
	printf("Max thread per block: %d\n", prop.maxThreadsPerBlock);
	printf("Max grid size: %d, %d, %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("Max thread dim: %d, %d, %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	auto img = cv::imread("./inputs/1.jpeg");
	printf("Image size: %dx%d\n", img.cols, img.rows);
	printf("Data type: %d, data size: %ld, warpSize: \n", img.type(), img.elemSize());
	u8* img_cuda = nullptr;
	hipMalloc(&img_cuda, img.cols * img.rows * img.elemSize());
	auto ret = hipMemcpy(img_cuda, img.ptr(), img.cols * img.rows * img.elemSize(), hipMemcpyHostToDevice);
	if (ret == hipSuccess)
	{
		printf("Image data copy success\n");
	}
	// kernel<<<1, 1>>>();
	hipDeviceSynchronize();
	printf("Hello from CUDA kernel!\n");
	float *d_A, *d_B, *d_C;
	hipMalloc(&d_A, 1024 * 1024 * 1024);
	hipMalloc(&d_B, 512);
	hipMalloc(&d_C, 128);
	return 0;
}


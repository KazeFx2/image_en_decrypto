#include "hip/hip_runtime.h"
//
// Created by kazefx on 25-2-17.
//

#include "private/Cuda.cuh"
#ifdef _WIN32
#define _USE_MATH_DEFINES
#include <math.h>
#endif

#define THREADS_PER_BLOCK 32

void* AllocCopyMatToCuda(__IN const cv::Mat& srcImage)
{
    void* p = MallocCuda(srcImage.rows * srcImage.cols * srcImage.elemSize());
    if (p != nullptr)
        hipMemcpy(p, srcImage.data, srcImage.rows * srcImage.cols * srcImage.elemSize(), hipMemcpyHostToDevice);
    return p;
}

void CopyMatToCuda(void* dstImage, const cv::Mat& srcImage)
{
    if (dstImage != nullptr)
        hipMemcpy(dstImage, srcImage.data, srcImage.rows * srcImage.cols * srcImage.elemSize(),
                   hipMemcpyHostToDevice);
}

void CopyCudaToMat(const cv::Mat& dstImage, const void* srcImage)
{
    hipMemcpy(dstImage.data, srcImage, dstImage.rows * dstImage.cols * dstImage.elemSize(), hipMemcpyDeviceToHost);
}


void* MallocCuda(__IN const u32 size)
{
    void* p = nullptr;
    hipError_t err = hipMalloc(&p, size);
    if (err != hipSuccess)
    {
        printf("CudaMalloc failed, %s\n", hipGetErrorString(err));
    }
    return p;
}

void FreeCuda(__IN void* ptr)
{
    if (ptr != nullptr)
        hipFree(ptr);
}

__device__ void ConfusionFuncCuda(__IN const u32 row, __IN const u32 col, __IN const u32 width, __IN const u32 height,
                                  __IN const u32 confusionSeed,
                                  __OUT u32& newRow,
                                  __OUT u32& newCol)
{
    newRow = (row + col) % height;
    const u32 tmp = static_cast<u32>(static_cast<i32>(rint(confusionSeed * sin(2 * M_PI * newRow / height)))) % width;
    newCol = (col + tmp) % width;
}

__global__ void DoConfusionCuda(__IN u8* dst, __IN const u8* src, __IN const u32 confusionSeed, __IN const u32 width,
                                __IN const u32 height, __IN const u8 nChannel)
{
    const u32 blockId = blockIdx.x + blockIdx.y * gridDim.x;
    u32 idx = threadIdx.x + blockId * blockDim.x;
    const u32 r = idx / width;
    const u32 c = idx % width;
    if (r < height && c < width)
    {
        u32 nr, nc;
        ConfusionFuncCuda(r, c, width, height, confusionSeed, nr, nc);
        u32 newIdx = (nr * width + nc) * nChannel;
        idx *= nChannel;
        u8 times = nChannel;
        while (times--)
            dst[newIdx++] = src[idx++];
    }
}

void ConfusionCuda(__OUT void* dstImage, __IN const void* srcImage,
                   __IN const cv::Size& size, __IN const u32 confusionSeed, __IN const u8 nChannel)
{
    const u32 fa = static_cast<u32>(sqrt(THREADS_PER_BLOCK));
    DoConfusionCuda<<<dim3((size.width + fa - 1) / fa,
                           (size.height) / fa
    ), THREADS_PER_BLOCK>>>(static_cast<u8*>(dstImage), static_cast<const u8*>(srcImage),
                            confusionSeed, size.width,
                            size.height, nChannel);
}


__device__ void InvertConfusionFuncCuda(__IN const u32 row, __IN const u32 col, __IN const u32 width,
                                        __IN const u32 height,
                                        __IN const u32 confusionSeed,
                                        __OUT u32& newRow,
                                        __OUT u32& newCol)
{
    const u32 tmp = static_cast<u32>(static_cast<i32>(rint(confusionSeed * sin(2 * M_PI * row / height)))) % width;
    newCol = (col + width - tmp) % width;
    newRow = (row + height - newCol % height) % height;
}

__global__ void DoInvertConfusionCuda(__IN u8* dst, __IN const u8* src, __IN const u32 confusionSeed,
                                      __IN const u32 width,
                                      __IN const u32 height, __IN const u8 nChannel)
{
    const u32 blockId = blockIdx.x + blockIdx.y * gridDim.x;
    u32 idx = threadIdx.x + blockId * blockDim.x;
    const u32 r = idx / width;
    const u32 c = idx % width;
    if (r < height && c < width)
    {
        u32 nr, nc;
        InvertConfusionFuncCuda(r, c, width, height, confusionSeed, nr, nc);
        u32 newIdx = (nr * width + nc) * nChannel;
        idx *= nChannel;
        u8 times = nChannel;
        while (times--)
            dst[newIdx++] = src[idx++];
    }
}

void InvertConfusionCuda(__OUT void* dstImage, __IN const void* srcImage,
                         __IN const cv::Size& size, __IN const u32 confusionSeed, __IN const u8 nChannel)
{
    const u32 fa = static_cast<u32>(sqrt(THREADS_PER_BLOCK));
    DoInvertConfusionCuda<<<dim3((size.width + fa - 1) / fa,
                                 (size.height + fa - 1) / fa
    ), THREADS_PER_BLOCK>>>(static_cast<u8*>(dstImage), static_cast<const u8*>(srcImage),
                            confusionSeed, size.width,
                            size.height, nChannel);
}

bool CudaAvailable()
{
    int deviceCount = 0;
    const hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess || deviceCount <= 0)
    {
        return false;
    }
    return true;
}

